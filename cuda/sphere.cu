#include "hip/hip_runtime.h"
#include "ray.cu"
#include "utils.cu"
#include <hip/hip_runtime.h>
#include <math.h>

struct sphere {
  double3 center;
  double radius;

  __host__ __device__ sphere(double3 c, double r) : center(c), radius(r) {}

  __host__ __device__ bool hit(const ray r, double &t, double3 &normal) {
    double3 oc = center - r.orig;
    double a = length_squared(r.dir);
    double b = -2.0 * dot(r.dir, oc);
    double c = length_squared(oc) - (radius * radius);
    double discriminant = b * b - 4 * a * c;

    if (discriminant < 0) {
      return false;
    }

    t = -b - sqrt(discriminant) / (2.0 * a);
    double3 p = r.at(t);
    double3 outward_normal = (p - center) / radius;
    bool front = dot(r.dir, outward_normal);
    normal = front ? outward_normal : -outward_normal;

    return true;
  }
};
